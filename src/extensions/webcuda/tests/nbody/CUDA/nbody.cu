#include <stdio.h>

#include <hip/hip_runtime.h>

#define NUMBODIES 1024
#define TIMESTEP 0.01 
#define NUMITERATIONS 40
#define NUMTHREADS 16
#define NUMBLOCKS 8

	__device__ float3
bodyBodyInteraction(float4 bi, float4 bj, float3 ai)
{
	float3 r;

	// r_ij  [3 FLOPS]
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	r.z = bj.z - bi.z;

	// distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
	float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
	//need this for the case of comparing the point to itself
	if(distSqr == 0){
		return ai;
	}

	// invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
	float distSixth = distSqr * distSqr * distSqr;
	float invDistCube =  1.0f/sqrtf(distSixth);

	// s = m_j * invDistCube [1 FLOP]
	float s = bj.w * invDistCube;

	// a_i =  a_i + s * r_ij [6 FLOPS]
	ai.x += r.x * s;
	ai.y += r.y * s;
	ai.z += r.z * s;

	return ai;
}

	__device__ float3
tile_calculation(float4 myPosition, float3 accel)
{
	int i;
	extern __shared__ float4 sharedPos[];
	for (i = 0; i < blockDim.x; i++) {
		accel = bodyBodyInteraction(myPosition, sharedPos[i], accel);
	}
	return accel;
}

	__global__ void
calculate_forces(void *devX, void *devV, int num_bodies, int num_iterations, float timestep)
{
	//typename vec4<T>::Type *sharedPos = SharedMemory<typename vec4<T>::Type>();
	extern __shared__ float4 sharedPos[];

	float4 *globalX = (float4 *)devX;
	float3 *globalV = (float3 *)devV;
	float4 myPosition;
	//overriding for time being
	int i, j,  tile;

	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	
	for(i = 0; i < num_iterations; i++){
		//have to reset acceleration before each iteration
		float3 acc = {0.0f, 0.0f, 0.0f};

		myPosition = globalX[gtid];
		//calculating new acceleration
		for (j = 0, tile = 0; j < num_bodies; j += blockDim.x , tile++)
		{
			int idx = tile * blockDim.x + threadIdx.x;
			sharedPos[threadIdx.x] = globalX[idx];

			//cannot use shared memory before all threads have put in proper value
			__syncthreads();

			// This is the "tile_calculation" from the GPUG3 article.
			acc = tile_calculation(myPosition, acc);

			__syncthreads();
		}
		//modifying velocity
		globalV[gtid].x += timestep*acc.x;
		globalV[gtid].y += timestep*acc.y;
		globalV[gtid].z += timestep*acc.z;

		//modifying position
		globalX[gtid].x += timestep*globalV[gtid].x;
		globalX[gtid].y += timestep*globalV[gtid].y;
		globalX[gtid].z += timestep*globalV[gtid].z;
		//need to wait for all thread to calculate new result before continuing
		__syncthreads();
	}


}

void check_CUDA_op(hipError_t error, char * message)
{
	if (error != hipSuccess)
	{
		fprintf(stderr, "%s", message);
		fprintf(stderr, " (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

}

void read_file(char *pathname, int length, float4 *position, float3 *velocity){
	FILE *fp;
	fp = fopen(pathname, "r");
	int i = 0;
	while(fscanf(fp, "%f %f %f %f %f %f %f", 
				&(position[i].w),
				&(position[i].x),
				&(position[i].y),
				&(position[i].z),
				&(velocity[i].x),
				&(velocity[i].y),
				&(velocity[i].z)) != EOF){
		i++;
	}
	printf("number of rows found: %d\n", i);

	fclose(fp);
}

void write_file(char *pathname, int length, float4 *position, float3 *velocity){
	FILE *fp;
	fp = fopen(pathname, "w");
	for(int i=0; i < length; i++){
		fprintf(fp, "%f %f %f %f",
				position[i].w, position[i].x, position[i].y, position[i].z);
		fprintf(fp, " %f %f %f\n",
				velocity[i].x, velocity[i].y, velocity[i].z);
	}
	printf("done writing file\n");

	fclose(fp);
}

int main(void)
{
	dim3 blocks, threads;

	char * pathname = "../data/tab1024";

    hipEvent_t start_event, stop_event;
    int eventflags = hipEventDefault;
    
    float hostMemAlloc, deviceMemAlloc, memcpyHtoD, memcpyDtoH, kernel, hostMemFree, deviceMemFree;
    hipEventCreateWithFlags(&start_event, eventflags);
    hipEventCreateWithFlags(&stop_event, eventflags);

	//allocate host memory for position, velocity
    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
	float4 *h_X = (float4 *) malloc(sizeof(float4) * NUMBODIES);
	float3 *h_V = (float3 *) malloc(sizeof(float3) * NUMBODIES);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&hostMemAlloc, start_event, stop_event);




	read_file(pathname, NUMBODIES, h_X, h_V);


	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;


	// Allocate the device input image 
	float4 *d_X = NULL;
	float3 *d_V = NULL;


    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed


	err = hipMalloc((void **)&d_X, NUMBODIES*sizeof(float4));
	err = hipMalloc((void **)&d_V, NUMBODIES*sizeof(float3));

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&deviceMemAlloc, start_event, stop_event);

	check_CUDA_op(err, "Failed to allocate device memory for vector array");

	//Copy position, vector data over to kernel
	printf("Copy position data from the CUDA device to the host memory\n");

    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
	err = hipMemcpy(d_X, h_X, NUMBODIES*sizeof(float4), hipMemcpyHostToDevice);
	err = hipMemcpy(d_V, h_V, NUMBODIES*sizeof(float3), hipMemcpyHostToDevice);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&memcpyHtoD, start_event, stop_event);

	check_CUDA_op(err, "Failed to copy vector data to device");


	int shared_mem_size	= sizeof(float4)*NUMTHREADS;

	printf("shared mem size %d\n", shared_mem_size);
	// Launch the Vector Add CUDA Kernel
	blocks = dim3(NUMBLOCKS);
	threads = dim3(NUMTHREADS);
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocks.x, threads.x);

    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
	calculate_forces<<< blocks, threads, shared_mem_size >>>(d_X, d_V, NUMBODIES, NUMITERATIONS, TIMESTEP);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&kernel, start_event, stop_event);




	err = hipGetLastError();
	check_CUDA_op(err, "Failed to to launch calculate_forces kernel");


	printf("Copy output data from the CUDA device to the host memory\n");

   hipEventRecord(start_event, 0);
	err = hipMemcpy(h_X, d_X, NUMBODIES*sizeof(float4), hipMemcpyDeviceToHost);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event); 
    hipEventElapsedTime(&memcpyDtoH, start_event, stop_event);


	check_CUDA_op(err, "Failed to to launch calculate_forces kernel");

	// Free device global memory
    hipEventRecord(start_event, 0);   
	err = hipFree(d_X);
	err = hipFree(d_V);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&deviceMemFree, start_event, stop_event);

	check_CUDA_op(err, "Failed to free device velocity memory");

	//write results
	//write_file("temp.txt", NUMBODIES, h_X, h_V);

	// Free host memory

    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
	free(h_X);
	free(h_V);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&hostMemFree, start_event, stop_event);

    printf("Host Mem Alloc: %f\nDevice Mem Alloc: %f\nMem Copy H to D: %f\nKernel: %f\nMem Copy D to H: %f\nHost Mem Free: %f\nDevice Mem Free: %f\n\n",
            hostMemAlloc*1000, deviceMemAlloc*1000,memcpyHtoD*1000, kernel*1000,memcpyDtoH*1000, hostMemFree*1000, deviceMemFree*1000);


	// Reset the device and exit
	err = hipDeviceReset();
	check_CUDA_op(err, "Failed to deinitialize the device ");

	return 0;

}



