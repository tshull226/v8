/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

#include <stdio.h>
#include "bs_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    //float t = 0.5f;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//const int OPT_N = 4000000;
//const int NUM_ITERATIONS = 512;
const int OPT_N = 1024;
const int NUM_ITERATIONS = 1;


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    //'h_' prefix - CPU (host) memory space
    float
    //CPU copy of GPU results
    *h_CallResultGPU,
    *h_PutResultGPU,
    //CPU instance of input data
    *h_StockPrice,
    *h_OptionStrike,
    *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
    //Results calculated by GPU
    *d_CallResult,
    *d_PutResult,
    //GPU instance of input data
    *d_StockPrice,
    *d_OptionStrike,
    *d_OptionYears;

    float hostMemAlloc, deviceMemAlloc, memcpyHtoD, memcpyDtoH, kernel, hostMemFree, deviceMemFree;

    printf("Initializing data...\n");
    printf("...allocating CPU memory for options.\n");


    hipEvent_t start_event, stop_event;
    int eventflags = hipEventDefault;
    
    hipEventCreateWithFlags(&start_event, eventflags);
    hipEventCreateWithFlags(&stop_event, eventflags);

    hipEventRecord(start_event, 0);  

    h_CallResultGPU = (float *)malloc(OPT_SZ);
    h_PutResultGPU  = (float *)malloc(OPT_SZ);
    h_StockPrice    = (float *)malloc(OPT_SZ);
    h_OptionStrike  = (float *)malloc(OPT_SZ);
    h_OptionYears   = (float *)malloc(OPT_SZ);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);  
    hipEventElapsedTime(&hostMemAlloc, start_event, stop_event);

    printf("...allocating GPU memory for options.\n");

    hipEventRecord(start_event, 0); 

    hipMalloc((void **)&d_CallResult,   OPT_SZ);
    hipMalloc((void **)&d_PutResult,    OPT_SZ);
    hipMalloc((void **)&d_StockPrice,   OPT_SZ);
    hipMalloc((void **)&d_OptionStrike, OPT_SZ);
    hipMalloc((void **)&d_OptionYears,  OPT_SZ);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event); 
    hipEventElapsedTime(&deviceMemAlloc, start_event, stop_event);

    printf("...generating input data in CPU mem.\n");
    srand(5347);

    //Generate options set
    int i = 0;
    for (i = 0; i < OPT_N; i++)
    {
        h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
        h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
        h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
    }
/*
    for (i = 0; i < OPT_N; i++)
    {
        //printf("%f %f %f\n", h_StockPrice[i], h_OptionStrike[i], h_OptionYears[i]);
        printf("%f, ", h_OptionYears[i]);
    }
*/
    printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing

    hipEventRecord(start_event, 0);   

    hipMemcpy(d_StockPrice,  h_StockPrice,   OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionStrike, h_OptionStrike,  OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionYears,  h_OptionYears,   OPT_SZ, hipMemcpyHostToDevice);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   
    hipEventElapsedTime(&memcpyHtoD, start_event, stop_event);

    printf("Data init done.\n\n");


    printf("Executing Black-Scholes GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
    hipDeviceSynchronize();

    hipEventRecord(start_event, 0);     

    for (i = 0; i < NUM_ITERATIONS; i++)
    {
        BlackScholesGPU<<<DIV_UP(OPT_N, 128), 128/*480, 128*/>>>(
            d_CallResult,
            d_PutResult,
            d_StockPrice,
            d_OptionStrike,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            OPT_N
        );
        //getLastCudaError("BlackScholesGPU() execution failed\n");
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event); 
    hipEventElapsedTime(&kernel, start_event, stop_event);

    hipDeviceSynchronize();

    printf("\nReading back GPU results...\n");
    //Read back GPU results to compare them to CPU results
   hipEventRecord(start_event, 0);

    hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost);
    hipMemcpy(h_PutResultGPU,  d_PutResult,  OPT_SZ, hipMemcpyDeviceToHost);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event); 
    hipEventElapsedTime(&memcpyDtoH, start_event, stop_event);

    for (i = 0; i < OPT_N; i++)
    {
    //    printf("%f %f\n", h_CallResultGPU[i], h_PutResultGPU[i]);
    }

    printf("Shutting down...\n");
    printf("...releasing GPU memory.\n");

    hipEventRecord(start_event, 0); 

    hipFree(d_OptionYears);
    hipFree(d_OptionStrike);
    hipFree(d_StockPrice);
    hipFree(d_PutResult);
    hipFree(d_CallResult);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&deviceMemFree, start_event, stop_event);

    printf("...releasing CPU memory.\n");

    hipEventRecord(start_event, 0); 

    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_PutResultGPU);
    free(h_CallResultGPU);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&hostMemFree, start_event, stop_event);

    printf("Shutdown done.\n");

    printf("Host Mem Alloc: %f\nDevice Mem Alloc: %f\nMem Copy H to D: %f\nKernel: %f\nMem Copy D to H: %f\nHost Mem Free: %f\nDevice Mem Free: %f\n\n",
            hostMemAlloc*1000000, deviceMemAlloc*1000000,memcpyHtoD*1000000, kernel*1000000,memcpyDtoH*1000000, hostMemFree*1000000, deviceMemFree*1000000);

    hipDeviceReset();

    //exit(EXIT_SUCCESS);
    return 0;
}
