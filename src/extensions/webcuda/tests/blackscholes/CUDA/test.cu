/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

#include <stdio.h>
#include "bs_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    //float t = 0.5f;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//const int OPT_N = 4000000;
//const int NUM_ITERATIONS = 512;
const int OPT_N = 1024;
const int NUM_ITERATIONS = 1;


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    //'h_' prefix - CPU (host) memory space
    float
    //CPU copy of GPU results
    *h_CallResultGPU,
    *h_PutResultGPU,
    //CPU instance of input data
    *h_StockPrice,
    *h_OptionStrike,
    *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
    //Results calculated by GPU
    *d_CallResult,
    *d_PutResult,
    //GPU instance of input data
    *d_StockPrice,
    *d_OptionStrike,
    *d_OptionYears;

    printf("Initializing data...\n");
    printf("...allocating CPU memory for options.\n");
    h_CallResultGPU = (float *)malloc(OPT_SZ);
    h_PutResultGPU  = (float *)malloc(OPT_SZ);
    h_StockPrice    = (float *)malloc(OPT_SZ);
    h_OptionStrike  = (float *)malloc(OPT_SZ);
    h_OptionYears   = (float *)malloc(OPT_SZ);

    printf("...allocating GPU memory for options.\n");
    hipMalloc((void **)&d_CallResult,   OPT_SZ);
    hipMalloc((void **)&d_PutResult,    OPT_SZ);
    hipMalloc((void **)&d_StockPrice,   OPT_SZ);
    hipMalloc((void **)&d_OptionStrike, OPT_SZ);
    hipMalloc((void **)&d_OptionYears,  OPT_SZ);

    printf("...generating input data in CPU mem.\n");
    srand(5347);

    //Generate options set
    int i = 0;
    for (i = 0; i < OPT_N; i++)
    {
        h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
        h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
        h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
    }
/*
    for (i = 0; i < OPT_N; i++)
    {
        //printf("%f %f %f\n", h_StockPrice[i], h_OptionStrike[i], h_OptionYears[i]);
        printf("%f, ", h_OptionYears[i]);
    }
*/
    printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing
    hipMemcpy(d_StockPrice,  h_StockPrice,   OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionStrike, h_OptionStrike,  OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionYears,  h_OptionYears,   OPT_SZ, hipMemcpyHostToDevice);
    printf("Data init done.\n\n");


    printf("Executing Black-Scholes GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
    hipDeviceSynchronize();

    for (i = 0; i < NUM_ITERATIONS; i++)
    {
        BlackScholesGPU<<<DIV_UP(OPT_N, 128), 128/*480, 128*/>>>(
            d_CallResult,
            d_PutResult,
            d_StockPrice,
            d_OptionStrike,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            OPT_N
        );
        //getLastCudaError("BlackScholesGPU() execution failed\n");
    }

    hipDeviceSynchronize();

    printf("\nReading back GPU results...\n");
    //Read back GPU results to compare them to CPU results
    hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost);
    hipMemcpy(h_PutResultGPU,  d_PutResult,  OPT_SZ, hipMemcpyDeviceToHost);

    for (i = 0; i < OPT_N; i++)
    {
        printf("%f %f\n", h_CallResultGPU[i], h_PutResultGPU[i]);
    }

    printf("Shutting down...\n");
    printf("...releasing GPU memory.\n");
    hipFree(d_OptionYears);
    hipFree(d_OptionStrike);
    hipFree(d_StockPrice);
    hipFree(d_PutResult);
    hipFree(d_CallResult);

    printf("...releasing CPU memory.\n");
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_PutResultGPU);
    free(h_CallResultGPU);
    printf("Shutdown done.\n");

    hipDeviceReset();

    //exit(EXIT_SUCCESS);
    return 0;
}
